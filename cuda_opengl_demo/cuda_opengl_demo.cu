#include "hip/hip_runtime.h"
 

// Utilities and system includes

#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    // ����u v ����
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    // ��������Ҳ�ģʽ
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    // write output vertex
    pos[y * width + x] = make_float4(u, w, v, 1.0f);
}


 void launch_kernel(float4* pos, unsigned int mesh_width,
    unsigned int mesh_height, float time)
{
    // execute the kernel  �߳̿�ߴ�
    dim3 block(8, 8, 1);
    // ����ߴ�
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    // <<<�߳̿飬 ���ٸ��߳�>>>
    simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}


